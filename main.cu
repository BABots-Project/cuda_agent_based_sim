#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include "include/json.hpp"
#include <fstream>
#include <iostream>
#include "headers/parameters.h"
#include "headers/init_env.h"
#include "headers/agent_update.h"
#include "headers/update_matrices.h"
#include "headers/logging.h"


int main(int argc, char* argv[]) {
    // Create CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start, 0);
    float attractant_pheromone_strength = ATTRACTANT_PHEROMONE_STRENGTH, repulsive_pheromone_strength = REPULSIVE_PHEROMONE_STRENGTH, odor_strength = ATTRACTION_STRENGTH, sigma = SIGMA, environmental_noise = ENVIRONMENTAL_NOISE;
    float* grid, * h_grid = new float[N * N], * attractive_pheromone, * repulsive_pheromone, * h_attractive_pheromone = new float[N * N];
    float* h_repulsive_pheromone = new float[N * N], * h_potential = new float[N * N], * potential;
    int worm_count = WORM_COUNT, exp_number = 0, * agent_count_grid, * h_agent_count_grid = new int[N * N];;


    printf("Found %d arguments\n", argc-1);
    switch (argc-1) {
        case 2:
            if(std::isdigit(argv[1][0]) && std::isdigit(argv[2][0])){
                /*attractant_pheromone_strength = std::stof(argv[1]);
                printf("Attractant pheromone strength: %.10f\n", attractant_pheromone_strength);
                repulsive_pheromone_strength = std::stof(argv[2]);
                printf("Repulsive pheromone strength: %.10f\n", repulsive_pheromone_strength);*/
                sigma = std::stof(argv[1]);
                printf("Sigma: %.10f\n", sigma);
                environmental_noise = std::stof(argv[2]);
                printf("Environmental noise: %.10f\n", environmental_noise);
            }
            break;

        case 5:
            if(std::isdigit(argv[1][0]) && std::isdigit(argv[2][0]) && std::isdigit(argv[3][0]) && std::isdigit(argv[4][0])  && std::isdigit(argv[5][0])){
                exp_number = std::stoi(argv[1]);
                printf("Experiment number: %d\n", exp_number);
                worm_count = std::stoi(argv[2]);
                printf("Worm count: %d\n", worm_count);
                attractant_pheromone_strength = std::stof(argv[3]);
                printf("Attractant pheromone strength: %.10f\n", attractant_pheromone_strength);
                repulsive_pheromone_strength = -std::stof(argv[4]);
                printf("Repulsive pheromone strength: %.10f\n", repulsive_pheromone_strength);
                int using_odor = std::stoi(argv[5]);
                if(using_odor == 0){
                    odor_strength = 0.0f;
                }
                printf("Odor strength: %f\n", odor_strength);
            }
            break;
        case 0:
            printf("No input arguments provided.\n");
            break;
    }
    Agent* d_agents, *h_agents = new Agent[worm_count];
    hiprandState* d_states, *d_states_grids;
    bool broken = false;
    size_t size = worm_count * sizeof(Agent);

    auto* positions = new float[worm_count * N_STEPS * 2]; // Matrix to store positions (x, y) for each agent at each timestep
    auto* angles = new float[worm_count * N_STEPS]; // Matrix to store angles for each agent at each timestep
    auto* velocities = new float[worm_count * N_STEPS]; // Matrix to store velocities for each agent at each timestep
    hipMalloc(&d_agents, size);
    hipMalloc(&d_states, worm_count * sizeof(hiprandState));
    hipMalloc(&d_states_grids, N * N * sizeof(hiprandState));
    hipMalloc(&grid, N*N*sizeof(float));
    hipMalloc(&potential, N*N*sizeof(float));

    // Initialize agent positions and random states
    initAgents<<<(worm_count + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_agents, d_states, time(NULL), worm_count);
    printf("Initializing agents\n");

    hipDeviceSynchronize();
    hipMemcpy(h_agents, d_agents, size, hipMemcpyDeviceToHost);

    dim3 gridSize((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);

    //initialize the agent count grid
    hipMalloc(&agent_count_grid, N*N*sizeof(int));
    initAgentDensityGrid<<<gridSize, blockSize>>>(agent_count_grid, d_agents, worm_count);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error in initAgentDensityGrid: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();
    hipMemcpy(h_agent_count_grid, agent_count_grid, N * N * sizeof(int), hipMemcpyDeviceToHost);

// Initialize the chemical grid concentration
    initGrid<<<gridSize, blockSize>>>(grid, d_states_grids);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error in initGrid: %s\n", hipGetErrorString(err));
    }

    hipDeviceSynchronize();
    hipMemcpy(h_grid, grid, N * N * sizeof(float), hipMemcpyDeviceToHost);

    //initialize the pheromone grids
    hipMalloc(&attractive_pheromone, N*N*sizeof(float));
    hipMalloc(&repulsive_pheromone, N*N*sizeof(float));
    initAttractiveAndRepulsivePheromoneGrid<<<gridSize, blockSize>>>(attractive_pheromone, repulsive_pheromone, agent_count_grid);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error in initAttractiveAndRepulsivePheromoneGrid: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();
    hipMemcpy(h_attractive_pheromone, attractive_pheromone, N * N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_repulsive_pheromone, repulsive_pheromone, N * N * sizeof(float), hipMemcpyDeviceToHost);

    //initialise the potential grid
    updatePotential<<<gridSize, blockSize>>>(potential, grid, attractive_pheromone, repulsive_pheromone, attractant_pheromone_strength, repulsive_pheromone_strength, odor_strength, d_states_grids, environmental_noise);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error in updatePotential: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();
    hipMemcpy(h_potential, potential, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Move agents in a loop
    for (int i = 0; i < N_STEPS; ++i) {
        //printf("Step %d\n", i);

        //copy the agent count grid to the device
        hipMemcpy(agent_count_grid, h_agent_count_grid, N * N * sizeof(int), hipMemcpyHostToDevice);
        moveAgents<<<(worm_count + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_agents, d_states, potential, agent_count_grid, worm_count, i, sigma);
        // Check for errors in the kernel launch
        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(err));
        }
        hipDeviceSynchronize();

        // Copy data from device to host
        hipMemcpy(h_agents, d_agents, size, hipMemcpyDeviceToHost);
        hipMemcpy(h_agent_count_grid, agent_count_grid, N * N * sizeof(int), hipMemcpyDeviceToHost);

        // Store positions, velocities and angles in the matrices
        for (int j = 0; j < worm_count; ++j) {
            positions[(i * worm_count + j) * 2] = h_agents[j].x;
            positions[(i * worm_count + j) * 2 + 1] = h_agents[j].y;

            angles[i * worm_count + j] = h_agents[j].angle;

            velocities[i * worm_count + j] = h_agents[j].speed;
        }

        //copy the repulsive pheromone grid to the device
        hipMemcpy(attractive_pheromone, h_attractive_pheromone, N * N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(repulsive_pheromone, h_repulsive_pheromone, N * N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(grid, h_grid, N * N * sizeof(float), hipMemcpyHostToDevice);

        //update all grids
        updateGrids<<<gridSize, blockSize>>>(grid, attractive_pheromone, repulsive_pheromone, agent_count_grid);
        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA error in updateGrids: %s\n", hipGetErrorString(err));
        }
        hipDeviceSynchronize();
        // copy data from device to host
        hipMemcpy(h_grid, grid, N * N * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(h_attractive_pheromone, attractive_pheromone, N * N * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(h_repulsive_pheromone, repulsive_pheromone, N * N * sizeof(float), hipMemcpyDeviceToHost);

        //update potential
        updatePotential<<<gridSize, blockSize>>>(potential, grid, attractive_pheromone, repulsive_pheromone, attractant_pheromone_strength, repulsive_pheromone_strength, odor_strength, d_states_grids, environmental_noise);
        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA error in updatePotential: %s\n", hipGetErrorString(err));
        }
        hipDeviceSynchronize();
        hipMemcpy(h_potential, potential, N * N * sizeof(float), hipMemcpyDeviceToHost);

        //check if any value in grid is invalid
        if (DEBUG){
            for (int i = 0; i < N; ++i) {
                for (int j = 0; j < N; ++j) {
                    if (isnan(h_grid[i * N + j]) || isinf(h_grid[i * N + j])) {
                        printf("Invalid concentration %f at (%d, %d)\n", h_grid[i * N + j], i, j);
                        broken = true;
                        break;
                    }
                    if (isnan(h_attractive_pheromone[i * N + j]) || isinf(h_attractive_pheromone[i * N + j])) {
                        printf("Invalid attractive pheromone %f at (%d, %d)\n", h_attractive_pheromone[i * N + j], i, j);
                        broken = true;
                        break;
                    }
                    if (isnan(h_repulsive_pheromone[i * N + j]) || isinf(h_repulsive_pheromone[i * N + j])) {
                        printf("Invalid repulsive pheromone %f at (%d, %d)\n", h_repulsive_pheromone[i * N + j], i, j);
                        broken = true;
                        break;
                    }
                    if (isnan(h_potential[i * N + j]) || isinf(h_potential[i * N + j])) {
                        printf("Invalid potential %f at (%d, %d)\n", h_potential[i * N + j], i, j);
                        broken = true;
                        break;
                    }
                }
            }
        }
        if (broken) {
            break;
        }
        // Save positions to JSON every LOGGING_INTERVAL steps
        if (i % LOGGING_INTERVAL == 0) {
            if(LOG_POTENTIAL) {
                logMatrixToFile("/home/nema/CLionProjects/untitled/logs/potential/potential_step_", h_potential, N, N, i);
            }
            if(LOG_AGENT_COUNT_GRID) {
                logIntMatrixToFile("/home/nema/CLionProjects/untitled/logs/agent_count/agents_log_step_", h_agent_count_grid, N, N, i);
            }
            if(LOG_GRID) {
                logMatrixToFile("/home/nema/CLionProjects/untitled/logs/chemical_concentration/chemical_concentration_step_", h_grid, N, N, i);
            }
            if(LOG_PHEROMONES) {
                logMatrixToFile("/home/nema/CLionProjects/untitled/logs/attractive_pheromone/attractive_pheromone_step_", h_attractive_pheromone, N, N, i);
                logMatrixToFile("/home/nema/CLionProjects/untitled/logs/repulsive_pheromone/repulsive_pheromone_step_", h_repulsive_pheromone, N, N, i);
            }

        }

    }
    if(LOG_GENERIC_TARGET_DATA) {
        saveAllDataToJSON("/home/nema/CLionProjects/untitled/agents_all_data.json", positions, velocities, angles, h_agents ,worm_count, N_STEPS);
    }

    /*if(LOG_TRAJECTORIES) {
        savePositionsToJSON("/home/nema/CLionProjects/untitled/agents_log.json", positions, worm_count, N_STEPS);
    }
    if(LOG_VELOCITIES) {
        savePositionsToJSON("/home/nema/CLionProjects/untitled/agents_velocities_log.json", velocities, worm_count, N_STEPS, true);
    }
    if(LOG_ANGLES) {
        savePositionsToJSON("/home/nema/CLionProjects/untitled/agents_angles_log.json", angles, worm_count, N_STEPS, true);
    } //
    saveInsideAreaToJSON("/home/nema/CLionProjects/untitled/inside_area.json", h_agents, worm_count, N_STEPS);*/
    hipFree(d_agents);
    hipFree(d_states);
    hipFree(grid);
    hipFree(potential);
    hipFree(attractive_pheromone);
    hipFree(repulsive_pheromone);
    hipFree(agent_count_grid);
    delete[] h_agents;
    delete[] h_grid;
    delete[] h_potential;
    delete[] h_attractive_pheromone;
    delete[] h_repulsive_pheromone;
    delete[] h_agent_count_grid;
    delete[] positions;
    delete[] angles;
    delete[] velocities;


    // Record the stop event
    hipEventRecord(stop, 0);

    // Synchronize the events
    hipEventSynchronize(stop);

    // Calculate the elapsed time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    // Print the elapsed time
    std::cout << "Elapsed time: " << elapsedTime << " ms" << std::endl;

    // Clean up
    hipEventDestroy(start);
    hipEventDestroy(stop);


    return 0;
}
