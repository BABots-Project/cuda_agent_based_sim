#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include "include/json.hpp"
#include <fstream>
#include <iostream>

using json = nlohmann::json;

#define N 128                        // Grid size
#define WORM_COUNT 500                 // Number of agents
#define WIDTH 20.0f          // Width of the 2D space
#define HEIGHT 20.0f         // Height of the 2D space
#define BLOCK_SIZE 32        // CUDA block size
#define N_STEPS 5000         // Number of simulation steps
#define LOGGING_INTERVAL 10    // Logging interval for saving positions
#define SPEED 0.015f            // Constant speed at which agents move
//#define DX WIDTH/N               // Grid spacing
#define LAMBDA 0.9f             //persistance of the movement
#define DRIFT_FACTOR 0.01f       //drift factor
#define SENSING_RANGE 1        //sensing range of the agents
#define MAX_CONCENTRATION 100.0f //maximum concentration of the chemical
#define DT 1.0f                //time step
#define GAMMA 0.0001f             //decay rate of the chemical
#define DIFFUSION_CONSTANT 0.001f                  //diffusion rate of the chemical
#define ATTRACTION_STRENGTH 0.111f
#define ATTRACTION_SCALE 1.5f
#define ODOR_THRESHOLD 0.1f
#define DEBUG false
#define SIGMA 0.001f
#define INITIAL_AREA_NUMBER_OF_CELLS 10 //defines the side length of the square where the agents are initialized in terms of number of cells
//pheromone parameters
#define ATTRACTANT_PHEROMONE_SCALE 1.5f
#define ATTRACTANT_PHEROMONE_STRENGTH 0.00411f
#define ATTRACTANT_PHEROMONE_DECAY_RATE 0.001f
#define ATTRACTANT_PHEROMONE_SECRETION_RATE 0.001f
#define ATTRACTANT_PHEROMONE_DIFFUSION_RATE 0.0001f
#define REPULSIVE_PHEROMONE_SCALE 1.5f
#define REPULSIVE_PHEROMONE_STRENGTH (-0.00000111f)
#define REPULSIVE_PHEROMONE_DECAY_RATE 0.001f
#define REPULSIVE_PHEROMONE_SECRETION_RATE 0.00001f
#define REPULSIVE_PHEROMONE_DIFFUSION_RATE 0.0001f
#define MAXIMUM_AGENTS_PER_CELL 4

__constant__ float DX = WIDTH/N;

struct Agent {
    float x, y, angle, speed;  // Position in 2D space
};

// Function to compute the gradient in the X direction (partial derivative)
__device__ float gradientX(float* grid, int i, int j) {
    //periodic boundary conditions
    int leftIndex = i - 1;
    if (leftIndex < 0) leftIndex += N;
    int rightIndex = i + 1;
    if (rightIndex >= N) rightIndex -= N;
    float left = grid[leftIndex * N + j];
    float right = grid[rightIndex * N + j];

    return (right - left) / (2.0f * DX);  // Central difference
}

// Function to compute the gradient in the Y direction (partial derivative)
__device__ float gradientY(float* grid, int i, int j) {
    int downIndex = j - 1;
    if (downIndex < 0) downIndex += N;
    int upIndex = j + 1;
    if (upIndex >= N) upIndex -= N;
    float down = grid[i * N + downIndex];
    float up = grid[i * N + upIndex];

    return (up - down) / (2.0f * DX);  // Central difference
}

// Function to compute the Laplacian (second derivative)
__device__ float laplacian(float* grid, int i, int j) {
    float center = grid[i * N + j];
    int leftIndex = i - 1;
    if (leftIndex < 0) leftIndex += N;
    int rightIndex = i + 1;
    if (rightIndex >= N) rightIndex -= N;
    int downIndex = j - 1;
    if (downIndex < 0) downIndex += N;
    int upIndex = j + 1;
    if (upIndex >= N) upIndex -= N;
    float left = grid[leftIndex * N + j];
    float right = grid[rightIndex * N + j];
    float down = grid[i * N + downIndex];
    float up = grid[i * N + upIndex];

    float laplacian = (left + right + up + down - 4.0f * center) / (DX * DX);
    if (isnan(laplacian) || isinf(laplacian)) {
        printf("Invalid laplacian %f at (%d, %d)\n", laplacian, i, j);
        printf("Center %f\n", center);
        printf("Left %f\n", left);
        printf("Right %f\n", right);
        printf("Down %f\n", down);
        printf("Up %f\n", up);
    }
    return laplacian;
}


// CUDA kernel to initialize the position of each agent
__global__ void initAgents(Agent* agents, hiprandState* states, unsigned long seed) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < WORM_COUNT) {
        hiprand_init(seed, id, 0, &states[id]);
        //agents[id].x = hiprand_uniform(&states[id]) * WIDTH;
        //agents[id].y = hiprand_uniform(&states[id]) * HEIGHT;
        //initialise in a random position inside the square centered at WIDTH/4, HEIGHT/4 with side length DX*INITIAL_AREA_NUMBER_OF_CELLS
        agents[id].x = WIDTH/4 + hiprand_uniform(&states[id]) * DX*INITIAL_AREA_NUMBER_OF_CELLS;
        agents[id].y = HEIGHT/4 + hiprand_uniform(&states[id]) * DX*INITIAL_AREA_NUMBER_OF_CELLS;
        agents[id].angle = hiprand_uniform(&states[id]) * 2 * M_PI;
        agents[id].speed = SPEED;
    }
}

// CUDA kernel to initialize the chemical grid concentration
__global__ void initGrid(float* grid) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i < N && j < N) {
        //place 100 units of chemical in the square in the middle of the grid with length 20
        if (i >= N / 2 - 10 && i < N / 2 + 10 && j >= N / 2 - 10 && j < N / 2 + 10) {
            grid[i * N + j] = MAX_CONCENTRATION;
        } else{
            grid[i * N + j] = 0.0f;
        }
    }
}



// CUDA kernel to initialize the pheromone grids
__global__ void initAttractiveAndRepulsivePheromoneGrid(float* attractive_pheromone, float* repulsive_pheromone, float* agent_density_grid) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i < N && j < N) {
        attractive_pheromone[i * N + j] = ATTRACTANT_PHEROMONE_SECRETION_RATE * ATTRACTANT_PHEROMONE_DECAY_RATE * agent_density_grid[i * N + j];
        repulsive_pheromone[i * N + j] = REPULSIVE_PHEROMONE_SECRETION_RATE * REPULSIVE_PHEROMONE_DECAY_RATE * agent_density_grid[i * N + j];
    }
}

//CUDA kernel to initialise the agent count grid
__global__ void initAgentDensityGrid(float* agent_count_grid, Agent* agents){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i < N && j < N) {
        agent_count_grid[i * N + j] = 0.0f;
        for (int k = 0; k < WORM_COUNT; ++k) {
            int agent_x = (int)(agents[k].x / DX);
            int agent_y = (int)(agents[k].y / DX);
            if (agent_x == i && agent_y == j) {
                agent_count_grid[i * N + j] += 1.0f;
            }
        }
    }
}

// CUDA kernel to update the position of each agent
__global__ void moveAgents(Agent* agents, hiprandState* states, float* grid, float*potential, float* agent_count_grid) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < WORM_COUNT) {
        //find the highest concentration of the chemical in the sensing range
        float max_concentration = 0.0f;
        int max_concentration_x = 0;
        int max_concentration_y = 0;

        int agent_x= (int)(agents[id].x / DX);
        int agent_y = (int)(agents[id].y / DX);
        for (int i = -SENSING_RANGE; i <= SENSING_RANGE; ++i) {
            for (int j = -SENSING_RANGE; j <= SENSING_RANGE; ++j) {
                float concentration = 0.0f;
                int xIndex = agent_x+i;
                int yIndex = agent_y+j;
                //apply periodic boundary conditions
                if (xIndex < 0) xIndex += N;
                if (xIndex >= N) xIndex -= N;
                if (yIndex < 0) yIndex += N;
                if (yIndex >= N) yIndex -= N;
                if (xIndex >= 0 && xIndex < N && yIndex >= 0 && yIndex < N) {
                    concentration = grid[xIndex * N + yIndex];
                }

                if (concentration > max_concentration) {
                    max_concentration = concentration;
                    max_concentration_x = i;
                    max_concentration_y = j;

                }
            }
        }
        float bias = atan2((float)max_concentration_y, (float)max_concentration_x );
        float random_angle = hiprand_uniform(&states[id]) * 2.0f * M_PI;
        float new_direction_x = cosf(random_angle)+(DRIFT_FACTOR * max_concentration*cosf(bias));
        float new_direction_y = sinf(random_angle)+(DRIFT_FACTOR * max_concentration*sinf(bias));
        float fx = LAMBDA * cosf(agents[id].angle) + (1.0f - LAMBDA) * new_direction_x;
        float fy = LAMBDA * sinf(agents[id].angle) + (1.0f - LAMBDA) * new_direction_y;
        float len = sqrt(fx * fx + fy * fy);
        fx /= len;
        fy /= len;
        float new_angle = atan2(fy, fx);
        float new_speed_x = SPEED + hiprand_uniform(&states[id]) * SIGMA;
        float new_speed_y = SPEED + hiprand_uniform(&states[id]) * SIGMA;
        if(max_concentration>ODOR_THRESHOLD){// || sensed_odor<ODOR_THRESHOLD){
            float potential_x = gradientX(potential, agent_x, agent_y);
            float potential_y = gradientY(potential, agent_x, agent_y);
            //printf("Potential x: %f, Potential y: %f\n", potential_x, potential_y);
            //printf("Sensed odor: %f\n", sensed_odor);
            new_speed_x =  abs(potential_x) + hiprand_uniform(&states[id]) * SIGMA;
            new_speed_y = abs(potential_y) + hiprand_uniform(&states[id]) * SIGMA;
            //printf("new speed x: %f, new speed y: %f\n", new_speed_x, new_speed_y);

        }

        float dx = fx * new_speed_x;
        float dy = fy * new_speed_y;


        agents[id].x += dx;
        agents[id].y += dy;
        agents[id].angle = new_angle;
        //agents[id].speed = new_speed;
        // Apply periodic boundary conditions
        if (agents[id].x < 0) agents[id].x += WIDTH;
        if (agents[id].x >= WIDTH) agents[id].x -= WIDTH;
        if (agents[id].y < 0) agents[id].y += HEIGHT;
        if (agents[id].y >= HEIGHT) agents[id].y -= HEIGHT;
        int new_x = (int)(agents[id].x / DX);
        int new_y = (int)(agents[id].y / DX);

        // Check if the new cell is full
        if (agent_count_grid[new_x * N + new_y] >= MAXIMUM_AGENTS_PER_CELL) {
            // Create an array of indices representing the neighboring cells
            int indices[] = {0, 1, 2, 3};
            // Shuffle the array of indices
            for (int k = MAXIMUM_AGENTS_PER_CELL-1; k > 0; --k) {
                int l = hiprand(&states[id]) % (k + 1);
                int tmp = indices[k];
                indices[k] = indices[l];
                indices[l] = tmp;
            }
            // Find a non-full neighboring cell
            int dx[] = {-1, 1, 0, 0};
            int dy[] = {0, 0, -1, 1};
            for (int k = 0; k < MAXIMUM_AGENTS_PER_CELL; ++k) {
                int nx = new_x + dx[indices[k]];
                int ny = new_y + dy[indices[k]];
                // Apply periodic boundary conditions
                if (nx < 0) nx += N;
                if (nx >= N) nx -= N;
                if (ny < 0) ny += N;
                if (ny >= N) ny -= N;
                // If the neighboring cell is not full, move the agent to this cell
                if (agent_count_grid[nx * N + ny] < MAXIMUM_AGENTS_PER_CELL) {
                    new_x = nx;
                    new_y = ny;
                    break;
                }
            }
        }

        // If the agent has moved to a new cell, update the agent_count_grid
        if (agent_x != new_x || agent_y != new_y) {
            // Decrease the count in the old cell
            atomicAdd(&agent_count_grid[agent_x * N + agent_y], -1);

            // Increase the count in the new cell
            atomicAdd(&agent_count_grid[new_x * N + new_y], 1);
        }
    }
}

//CUDA kernel to update all the grids (except the potential and the agent count grid)
__global__ void updateGrids(float* grid, float* attractive_pheromone, float* repulsive_pheromone, float* agent_count_grid){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i < N && j < N) {
        float laplacian_value = laplacian(grid, i, j);

        float new_concentration = grid[i * N + j] + DT * (DIFFUSION_CONSTANT * laplacian_value - GAMMA * grid[i * N + j]);
        if (new_concentration < 0) new_concentration = 0.0f;
        if (new_concentration > MAX_CONCENTRATION) new_concentration = MAX_CONCENTRATION;
        //check if the grid is a valid float number
        if (isnan(new_concentration) || isinf(new_concentration)) {
            printf("Invalid concentration %f at (%d, %d)\n", new_concentration, i, j);
            printf("Laplacian value %f\n", laplacian_value);
            printf("Old concentration %f\n", grid[i * N + j]);

        }

        grid[i * N + j] = new_concentration;

        //update attractive pheromone
        laplacian_value = laplacian(attractive_pheromone, i, j);
        float new_attractive_pheromone = attractive_pheromone[i * N + j] + DT * (ATTRACTANT_PHEROMONE_DIFFUSION_RATE * laplacian_value - ATTRACTANT_PHEROMONE_DECAY_RATE * attractive_pheromone[i * N + j] + ATTRACTANT_PHEROMONE_SECRETION_RATE * agent_count_grid[i * N + j] / (DX * DX));
        if (new_attractive_pheromone < 0) new_attractive_pheromone = 0.0f;
        attractive_pheromone[i * N + j] = new_attractive_pheromone;

        //update repulsive pheromone
        laplacian_value = laplacian(repulsive_pheromone, i, j);
        float new_repulsive_pheromone = repulsive_pheromone[i * N + j] + DT * (REPULSIVE_PHEROMONE_DIFFUSION_RATE * laplacian_value - REPULSIVE_PHEROMONE_DECAY_RATE * repulsive_pheromone[i * N + j] + REPULSIVE_PHEROMONE_SECRETION_RATE * agent_count_grid[i * N + j] / (DX * DX));
        if (new_repulsive_pheromone < 0) new_repulsive_pheromone = 0.0f;
        repulsive_pheromone[i * N + j] = new_repulsive_pheromone;
    }
}

//CUDA kernel to update the grid of the chemical concentration using a reaction-diffusion equation
__global__ void updateGrid(float* grid) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i < N && j < N) {

        float laplacian_value = laplacian(grid, i, j);

        float new_concentration = grid[i * N + j] + DT * (DIFFUSION_CONSTANT * laplacian_value - GAMMA * grid[i * N + j]);
        if (new_concentration < 0) new_concentration = 0.0f;
        if (new_concentration > MAX_CONCENTRATION) new_concentration = MAX_CONCENTRATION;
        //check if the grid is a valid float number
        if (isnan(new_concentration) || isinf(new_concentration)) {
            printf("Invalid concentration %f at (%d, %d)\n", new_concentration, i, j);
            printf("Laplacian value %f\n", laplacian_value);
            printf("Old concentration %f\n", grid[i * N + j]);

        }

        grid[i * N + j] = new_concentration;
    }
}

//CUDA kernel to update the potential matrix
__global__ void updatePotential(float* potential, float* grid, float* attractive_pheromone, float* repulsive_pheromone){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i < N && j < N) {
        float potential_odor, potential_attractive_pheromone, potential_repulsive_pheromone;
        potential_odor = ATTRACTION_STRENGTH * log10(ATTRACTION_SCALE + grid[i * N + j]);
        potential_attractive_pheromone = ATTRACTANT_PHEROMONE_STRENGTH * log10(ATTRACTANT_PHEROMONE_SCALE + attractive_pheromone[i * N + j]);
        potential_repulsive_pheromone = REPULSIVE_PHEROMONE_STRENGTH * log10(REPULSIVE_PHEROMONE_SCALE + repulsive_pheromone[i * N + j]);
        potential[i * N + j] = potential_odor + potential_attractive_pheromone + potential_repulsive_pheromone;
    }
}

// Function to save the positions of agents in a JSON file
void saveToJSON(const char* filename, Agent* h_agents, int step) {
    static json log;
    static bool initialized = false;

    if (!initialized) {
        // Log simulation parameters only once
        log["parameters"] = {{"WIDTH", WIDTH}, {"HEIGHT", HEIGHT}, {"N", WORM_COUNT}, {"LOGGING_INTERVAL", LOGGING_INTERVAL}, {"N_STEPS", N_STEPS} };
        initialized = true;
    }

    for (int i = 0; i < WORM_COUNT; ++i) {
        log[std::to_string(i)].push_back({ h_agents[i].x, h_agents[i].y });
    }

    std::ofstream outFile(filename);
    outFile << log.dump();  // Pretty-print JSON with an indentation of 4 spaces
    outFile.close();

}

// function to save the grid to a file
void saveGridToJSON(const char* filename, float* h_grid) {
    static json log;
    static bool initialized = false;

    if (!initialized) {
        // Log simulation parameters only once
        log["parameters"] = {{"WIDTH",            WIDTH},
                             {"HEIGHT",           HEIGHT},
                             {"N",                WORM_COUNT},
                             {"LOGGING_INTERVAL", LOGGING_INTERVAL},
                             {"N_STEPS",          N_STEPS}};
        initialized = true;
    }

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            //use (i, j) as the key for the JSON object
            log[std::to_string(i)+","+std::to_string(j)].push_back({h_grid[i * N + j]});
        }
    }

    std::ofstream outFile(filename);
    outFile << log.dump(4);  // Pretty-print JSON with an indentation of 4 spaces
    outFile.close();
}

// Function to log the matrix to a file
void logMatrixToFile(const char* filename, float* matrix, int width, int height, int step) {
    std::ofstream outFile(filename + std::to_string(step) + ".txt");
    if (!outFile) {
        std::cerr << "Error opening file for writing!" << std::endl;
        return;
    }

    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x) {
            outFile << matrix[y * width + x] << " ";
        }
        outFile << "\n";
    }

    outFile.close();
}

int main() {
    Agent* d_agents;
    Agent* h_agents = new Agent[WORM_COUNT];
    hiprandState* d_states;
    bool broken = false;
    size_t size = WORM_COUNT * sizeof(Agent);
    //float target_x = WIDTH / 2;
    //float target_y = HEIGHT / 2;
    float* grid;
    float* h_grid = new float[N * N];

    float* attractive_pheromone;
    float* repulsive_pheromone;
    float* agent_count_grid;
    float* h_attractive_pheromone = new float[N * N];
    float* h_repulsive_pheromone = new float[N * N];
    float* h_agent_count_grid = new float[N * N];


    float* h_potential = new float[N * N];
    float* potential;


    hipMalloc(&d_agents, size);
    hipMalloc(&d_states, WORM_COUNT * sizeof(hiprandState));
    hipMalloc(&grid, N*N*sizeof(float));
    hipMalloc(&potential, N*N*sizeof(float));

    // Initialize agent positions and random states
    initAgents<<<(WORM_COUNT + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_agents, d_states, time(NULL));
    printf("Initializing agents\n");

    hipDeviceSynchronize();
    hipMemcpy(h_agents, d_agents, size, hipMemcpyDeviceToHost);
    dim3 gridSize((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);

    //initialize the agent count grid
    hipMalloc(&agent_count_grid, N*N*sizeof(float));
    initAgentDensityGrid<<<gridSize, blockSize>>>(agent_count_grid, d_agents);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error in initAgentDensityGrid: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();
    hipMemcpy(h_agent_count_grid, agent_count_grid, N * N * sizeof(float), hipMemcpyDeviceToHost);


// Initialize the chemical grid concentration
    initGrid<<<gridSize, blockSize>>>(grid);

// Check for errors in the kernel launch
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error in initGrid: %s\n", hipGetErrorString(err));
    }

    hipDeviceSynchronize();
    hipMemcpy(h_grid, grid, N * N * sizeof(float), hipMemcpyDeviceToHost);

    //initialize the pheromone grids
    hipMalloc(&attractive_pheromone, N*N*sizeof(float));
    hipMalloc(&repulsive_pheromone, N*N*sizeof(float));
    initAttractiveAndRepulsivePheromoneGrid<<<gridSize, blockSize>>>(attractive_pheromone, repulsive_pheromone, agent_count_grid);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error in initAttractiveAndRepulsivePheromoneGrid: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();
    hipMemcpy(h_attractive_pheromone, attractive_pheromone, N * N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_repulsive_pheromone, repulsive_pheromone, N * N * sizeof(float), hipMemcpyDeviceToHost);

    updatePotential<<<gridSize, blockSize>>>(potential, grid, attractive_pheromone, repulsive_pheromone);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error in updatePotential: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();
    hipMemcpy(h_potential, potential, N * N * sizeof(float), hipMemcpyDeviceToHost);


    //print the grid:
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            if (h_grid[i * N + j] > 0) {
                printf("X ");
            } else {
                printf("  ");
            }

        }
        printf("\n");
    }

    // Move agents in a loop
    for (int i = 0; i < N_STEPS; ++i) {
        printf("Step %d\n", i);

        moveAgents<<<(WORM_COUNT + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_agents, d_states, grid, potential, agent_count_grid);
        // Check for errors in the kernel launch
        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(err));
        }
        hipDeviceSynchronize();

        // Copy data from device to host
        hipMemcpy(h_agents, d_agents, size, hipMemcpyDeviceToHost);
        hipMemcpy(h_agent_count_grid, agent_count_grid, N * N * sizeof(float), hipMemcpyDeviceToHost);

        //update all grids
        updateGrids<<<gridSize, blockSize>>>(grid, attractive_pheromone, repulsive_pheromone, agent_count_grid);
        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA error in updateGrids: %s\n", hipGetErrorString(err));
        }
        hipDeviceSynchronize();
        // copy data from device to host
        hipMemcpy(h_grid, grid, N * N * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(h_attractive_pheromone, attractive_pheromone, N * N * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(h_repulsive_pheromone, repulsive_pheromone, N * N * sizeof(float), hipMemcpyDeviceToHost);



        //update potential
        updatePotential<<<gridSize, blockSize>>>(potential, grid, attractive_pheromone, repulsive_pheromone);
        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA error in updatePotential: %s\n", hipGetErrorString(err));
        }
        hipDeviceSynchronize();
        hipMemcpy(h_potential, potential, N * N * sizeof(float), hipMemcpyDeviceToHost);

        //check if any value in grid is invalid
        if (DEBUG){
            for (int i = 0; i < N; ++i) {
                for (int j = 0; j < N; ++j) {
                    if (isnan(h_grid[i * N + j]) || isinf(h_grid[i * N + j])) {
                        printf("Invalid concentration %f at (%d, %d)\n", h_grid[i * N + j], i, j);
                        broken = true;
                        break;
                    }
                }
            }
        }
        if (broken) {
            break;
        }
        // Save positions to JSON every LOGGING_INTERVAL steps
        if (i % LOGGING_INTERVAL == 0) {
            saveToJSON("/home/nema/CLionProjects/untitled/agents_log.json", h_agents, i);
            //saveGridToJSON("/home/nema/CLionProjects/untitled/grid_log.json", h_grid);
            //saveGridToJSON("/home/nema/CLionProjects/untitled/agent_count_grid.json", h_agent_count_grid);
            logMatrixToFile("/home/nema/CLionProjects/untitled/logs/agent_count/agents_log_step_", h_agent_count_grid, N, N, i);
            logMatrixToFile("/home/nema/CLionProjects/untitled/logs/chemical_concentration/chemical_concentration_step_", h_grid, N, N, i);
            logMatrixToFile("/home/nema/CLionProjects/untitled/logs/attractive_pheromone/attractive_pheromone_step_", h_attractive_pheromone, N, N, i);
            logMatrixToFile("/home/nema/CLionProjects/untitled/logs/repulsive_pheromone/repulsive_pheromone_step_", h_repulsive_pheromone, N, N, i);
            logMatrixToFile("/home/nema/CLionProjects/untitled/logs/potential/potential_step_", h_potential, N, N, i);

        }
    }

    hipFree(d_agents);
    hipFree(d_states);
    hipFree(grid);
    hipFree(potential);
    hipFree(attractive_pheromone);
    hipFree(repulsive_pheromone);
    hipFree(agent_count_grid);
    delete[] h_agents;
    delete[] h_grid;
    delete[] h_potential;
    delete[] h_attractive_pheromone;
    delete[] h_repulsive_pheromone;
    delete[] h_agent_count_grid;
    return 0;
}
